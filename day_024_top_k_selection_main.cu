/*
 * 1 <= N <= 100,000,000
 * 1 <= k <= N
 * input is 32 bit values
 */


#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#ifdef LEET_GPU
#include "solve.h"
#include <cuda_runtime.h>
#endif

__device__ void CompareSwap(float *A, float *B, bool Dir)
{
    if ((*A > *B) == Dir)
    {
        float Temp = *A;
        *A = *B;
        *B = Temp;
    }
}

__global__ void SimpleSortKernel(float *Data, int N)
{
    for (int I = 0; I < N - 1; I++)
    {
        for (int J = 0; J < N - 1 - I; J++)
        {
            if (Data[J] < Data[J + 1])
            {
                float Temp = Data[J];
                Data[J] = Data[J + 1];
                Data[J + 1] = Temp;
            }
        }
    }
}

// input, output are device pointers
void Solve(const float *Input, float *Output, int N, int K)
{
    float *DData;
    hipError_t Err = hipMalloc(&DData, N * sizeof(float));
    if (Err != hipSuccess)
    {
        printf("CUDA malloc failed: %s\n", hipGetErrorString(Err));
        return;
    }

    Err = hipMemcpy(DData, Input, N * sizeof(float), hipMemcpyDeviceToDevice);
    if (Err != hipSuccess)
    {
        printf("CUDA memcpy failed: %s\n", hipGetErrorString(Err));
        hipFree(DData);
        return;
    }

    int BlockSize = 32;
    int GridSize = 1;

    SimpleSortKernel<<<GridSize, BlockSize>>>(DData, N);
    Err = hipGetLastError();
    if (Err != hipSuccess)
    {
        printf("Sort kernel failed: %s\n", hipGetErrorString(Err));
        hipFree(DData);
        return;
    }

    hipDeviceSynchronize();

    Err = hipMemcpy(Output, DData, K * sizeof(float), hipMemcpyDeviceToDevice);
    if (Err != hipSuccess)
    {
        printf("Output memcpy failed: %s\n", hipGetErrorString(Err));
    }

    hipFree(DData);
}

int main()
{
    const float H_INPUT[] = {1.0f, 5.0f, 3.0f, 2.0f, 4.0f};
    int N = 5;
    int K = 3;

    const float EXPECTED_OUTPUT[] = {5.0f, 4.0f, 3.0f};

    float *DInput;
    float *DOutput;
    float *HResult = (float *)malloc(K * sizeof(float));

    hipMalloc(&DInput, N * sizeof(float));
    hipMalloc(&DOutput, K * sizeof(float));

    hipMemcpy(DInput, H_INPUT, N * sizeof(float), hipMemcpyHostToDevice);

    Solve(DInput, DOutput, N, K);

    hipMemcpy(HResult, DOutput, K * sizeof(float), hipMemcpyDeviceToHost);

    printf("Input: ");
    for (int I = 0; I < N; I++)
    {
        printf("%.1f ", H_INPUT[I]);
    }
    printf("\n");

    printf("Top %d elements: ", K);
    for (int I = 0; I < K; I++)
    {
        printf("%.1f ", HResult[I]);
    }
    printf("\n");

    printf("Expected: ");
    for (int I = 0; I < K; I++)
    {
        printf("%.1f ", EXPECTED_OUTPUT[I]);
    }
    printf("\n");

    bool Correct = true;
    for (int I = 0; I < K; I++)
    {
        if (fabs(HResult[I] - EXPECTED_OUTPUT[I]) > 1e-6)
        {
            Correct = false;
            break;
        }
    }

    printf("Result: %s\n", Correct ? "PASS" : "FAIL");

    hipFree(DInput);
    hipFree(DOutput);
    free(HResult);

    return 0;
}
