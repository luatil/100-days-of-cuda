#include <stdio.h>

typedef unsigned int u32;
typedef float f32;

#include <stdio.h>

#include "day_001_macros.h"
#include "day_001_vector_add_kernel.cu"
#include "day_012_timed_cuda_block_002.cu"

int main(int ArgumentCount, char **Arguments)
{
    u32 ExitCode = 0;

    if (ArgumentCount == 2)
    {
        u32 N = 0;
        sscanf(Arguments[1], "%d", &N);

        u32 SizeInBytes = sizeof(f32) * N;

        f32 *HostA = AllocateCPU(f32, N);
        f32 *HostB = AllocateCPU(f32, N);
        f32 *HostC = AllocateCPU(f32, N);

        for (u32 I = 0; I < N; I++)
        {
            HostA[I] = 1.0f * I;
            HostB[I] = 2.0f * I;
        }

        f32 *DeviceA, *DeviceB, *DeviceC;

        // Allocations
        {
            hipMalloc(&DeviceA, SizeInBytes);
            hipMalloc(&DeviceB, SizeInBytes);
            hipMalloc(&DeviceC, SizeInBytes);
        }

        // Memcpy Host To Device
        {
            hipMemcpy(DeviceA, HostA, SizeInBytes, hipMemcpyHostToDevice);
            hipMemcpy(DeviceB, HostB, SizeInBytes, hipMemcpyHostToDevice);
        }

        // Kernel Launch
        {
            TimeCudaBlock("Add Kernel");
            u32 ThreadsPerBlock = 32;
            u32 BlocksPerGrid = (N + ThreadsPerBlock - 1) / ThreadsPerBlock;
            AddKernel<<<BlocksPerGrid, ThreadsPerBlock>>>(DeviceA, DeviceB, DeviceC, N);
        }

        // Kernel Launch
        {
            TimeCudaBandwidth("Add Kernel", 2 * SizeInBytes, SizeInBytes, 1.0f / (3.0f * 4.0f));
            u32 ThreadsPerBlock = 32;
            u32 BlocksPerGrid = (N + ThreadsPerBlock - 1) / ThreadsPerBlock;
            AddKernel<<<BlocksPerGrid, ThreadsPerBlock>>>(DeviceA, DeviceB, DeviceC, N);
        }

        // Memcpy Device To Host
        {
            hipMemcpy(HostC, DeviceC, SizeInBytes, hipMemcpyDeviceToHost);
        }

        fprintf(stdout, "First element: %f\n", HostC[0]);
        fprintf(stdout, "Last element: %f\n", HostC[N - 1]);
    }
    else
    {
        fprintf(stderr, "Usage: %s [number of vector elements]\n", Arguments[0]);
        ExitCode = 1;
    }

    return ExitCode;
}
