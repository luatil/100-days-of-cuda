/*
 * Day 008: Generate Random Matrix
 */
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <stdint.h>
#include <stdlib.h>

typedef float f32;
typedef uint32_t u32;
typedef uint64_t u64;

typedef int32_t s32;

__global__ void GenerateRandomMatrix(f32 *OutputMatrix, u32 Width, u32 Height, u64 Seed)
{
    u32 Tid = blockIdx.x * blockDim.x + threadIdx.x;
    u32 TotalElements = Width * Height;

    if (Tid < TotalElements)
    {
        hiprandState State;
        hiprand_init(Seed, Tid, 0, &State);
        OutputMatrix[Tid] = hiprand_uniform(&State); // Uniform [0,1]
    }
}
