
#include <hip/hip_runtime.h>
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>

#define F32_ABS(_X) fabs(_X)
#define ASSERT_CLOSE(_A, _B, _Tolerance)                                                                               \
    do                                                                                                                 \
    {                                                                                                                  \
        if (F32_ABS((_A) - (_B)) >= (_Tolerance))                                                                      \
        {                                                                                                              \
            printf("ASSERTION FAILED: %f != %f (diff: %f, tolerance: %f) at %s:%d\n", (float)(_A), (float)(_B),        \
                   (float)F32_ABS((_A) - (_B)), (float)(_Tolerance), __FILE__, __LINE__);                              \
            asm("trap;");                                                                                              \
        }                                                                                                              \
    } while (0)

__global__ void VectorAddKernel(float *A, float *B, float *C, int N)
{
    int Tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (Tid < N)
    {
        C[Tid] = A[Tid] + B[Tid];
    }
}

__global__ void CompareFloatArrayKernel(float *A, float *B, float Tolerance, int N)
{
    int Tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (Tid < N)
    {
        ASSERT_CLOSE(A[Tid], B[Tid], Tolerance);
    }
}

extern "C" void CompareFloatArray(float *A, float *B, float Tolerance, int N)
{
    int BlockDim = 256;
    int GridDim = (N + BlockDim - 1) / BlockDim;
    CompareFloatArrayKernel<<<GridDim, BlockDim>>>(A, B, Tolerance, N);

    hipError_t Err = hipDeviceSynchronize();
    if (Err != hipSuccess)
    {
        // printf("CUDA error during comparison: %s\n", cudaGetErrorString(Err));
        exit(1);
    }
}

extern "C" void VectorAdd(float *A, float *B, float *C, int N)
{
    int BlockDim = 256;
    int GridDim = (N + BlockDim - 1) / BlockDim;
    VectorAddKernel<<<GridDim, BlockDim>>>(A, B, C, N);
}

void TestVectorAdd(float *A, float *B, float *C, float *E, int N)
{
    float *DA, *DB, *DC, *DE;
    hipMalloc(&DA, sizeof(float) * N);
    hipMalloc(&DB, sizeof(float) * N);
    hipMalloc(&DC, sizeof(float) * N);
    hipMalloc(&DE, sizeof(float) * N);

    hipMemcpy(DA, A, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(DB, B, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(DC, C, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(DE, E, sizeof(float) * N, hipMemcpyHostToDevice);

    VectorAdd(DA, DB, DC, N);
    hipDeviceSynchronize();

    CompareFloatArray(DC, DE, 0.001, N);
    hipDeviceSynchronize();

    hipFree(DA);
    hipFree(DB);
    hipFree(DC);
    hipFree(DE);
}

void TestVectorAdd001()
{
    float A[] = {1.0f, 2.0f, 3.0f, 4.0f, 5.0f};
    float B[] = {5.0f, 4.0f, 3.0f, 2.0f, 1.0f};
    float C[] = {0.0f, 0.0f, 0.0f, 0.0f, 0.0f};
    float E[] = {6.0f, 6.0f, 6.0f, 6.0f, 6.0f};

    TestVectorAdd(A, B, C, E, 5);
}

int main()
{
    TestVectorAdd001();
}
