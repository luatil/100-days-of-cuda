#include "hip/hip_runtime.h"
#ifndef LEET_GPU_NO_IMPORT
#include "solve.h"
#include <hip/hip_runtime.h>
#endif

#define BLOCK_DIM 256
#define DEBUG 0
#define DEBUG_PARTIAL_SUMS 0

__global__ void CalculatePartialSums(const float *Input, float *Output, float *PartialSums, int N)
{
    __shared__ float Shared[BLOCK_DIM];

    // NOTE(luatil): Could do coarsening here
    int Tid = blockDim.x * blockIdx.x + threadIdx.x;
    int Tx = threadIdx.x;

    Shared[Tx] = Tid < N ? Input[Tid] : 0.0f;

    for (int Stride = 1; Stride <= blockDim.x / 2; Stride *= 2)
    {
        __syncthreads();
        if (Tx >= Stride)
        {
            Shared[Tx] += Shared[Tx - Stride];
        }
    }

#if DEBUG
    printf("Shared[%d]=%.3f\n", Tx, Shared[Tx]);
    printf("Input [%d]=%.3f\n", Tid, Input[Tid]);
#endif

    __syncthreads();
    if (Tid < N)
    {
        Output[Tid] = Shared[Tx];
    }

    int IsLastBlock = (blockDim.x * (blockIdx.x + 1)) > N;
    int IsLastElementOfInput = Tx == ((N - 1) - blockDim.x * blockIdx.x);
    int IsLastElementOfBlock = Tx == (blockDim.x - 1);

    if (IsLastBlock)
    {
        if (IsLastElementOfInput)
        {
            PartialSums[blockIdx.x] = Shared[Tx];
        }
    }
    else if (IsLastElementOfBlock)
    {
        PartialSums[blockIdx.x] = Shared[Tx];
    }
}

__global__ void ScanPartialSums(float *PartialSums, int N)
{
#if DEBUG_PARTIAL_SUMS
    for (int I = 0; I < N; I++)
    {
        printf("PartialSums[%d] = %.3f\n", I, PartialSums[I]);
    }
#endif
    for (int I = 1; I < N; I++)
    {
        PartialSums[I] += PartialSums[I - 1];
    }
#if DEBUG_PARTIAL_SUMS
    for (int I = 0; I < N; I++)
    {
        printf("PartialSums[%d] = %.3f\n", I, PartialSums[I]);
    }
#endif
}

__global__ void ExpandPartialSums(const float *PartialSums, float *Output, int N)
{
    const int IsFirstBlock = blockIdx.x == 0;
    const int Tid = blockDim.x * blockIdx.x + threadIdx.x;
    const int TidInRange = Tid < N;

    __shared__ float BlockPartialSum;

    if (!IsFirstBlock)
    {
        if (threadIdx.x == 0)
        {
            BlockPartialSum = PartialSums[blockIdx.x - 1];
        }
        __syncthreads();

        if (TidInRange)
        {
            Output[Tid] += BlockPartialSum;
        }
    }
}

// input, output are device pointers
void solve(const float *input, float *output, int N)
{
    const int NumberOfBlocks = (N + BLOCK_DIM - 1) / BLOCK_DIM;

    float *PartialSums;
    hipMalloc(&PartialSums, NumberOfBlocks * sizeof(float));

    CalculatePartialSums<<<NumberOfBlocks, BLOCK_DIM>>>(input, output, PartialSums, N);
    ScanPartialSums<<<1, 1>>>(PartialSums, NumberOfBlocks);
    // Could optimize by removing first block.
    ExpandPartialSums<<<NumberOfBlocks, BLOCK_DIM>>>(PartialSums, output, N);

    hipFree(PartialSums);
}
