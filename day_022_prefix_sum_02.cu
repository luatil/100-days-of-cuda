#include "hip/hip_runtime.h"
#ifndef LEET_GPU_NO_IMPORT
#include "solve.h"
#include <hip/hip_runtime.h>
#endif

#define BLOCK_DIM 256
#define DEBUG 0
#define DEBUG_PARTIAL_SUMS 0

__global__ void CalculatePartialSums(const float *Input, float *Output, float *PartialSums, int N)
{
    __shared__ float Shared[BLOCK_DIM];

    // NOTE(luatil): Could do coarsening here
    int Tid = blockDim.x * blockIdx.x + threadIdx.x;
    int Tx = threadIdx.x;

    Shared[Tx] = Tid < N ? Input[Tid] : 0.0f;
    __syncthreads();

    for (int Stride = 1; Stride <= blockDim.x / 2; Stride *= 2)
    {
        float Temp = 0.0f;
        if (Tx >= Stride)
        {
            Temp = Shared[Tx] + Shared[Tx - Stride];
        }
        __syncthreads();
        if (Tx >= Stride)
        {
            Shared[Tx] = Temp;
        }
        __syncthreads();
    }

#if DEBUG
    printf("Shared[%d]=%.3f\n", Tx, Shared[Tx]);
    printf("Input [%d]=%.3f\n", Tid, Input[Tid]);
#endif

    if (Tid < N)
    {
        Output[Tid] = Shared[Tx];
    }

    if (Tx == BLOCK_DIM - 1)
    {
        PartialSums[blockIdx.x] = Shared[Tx];
    }
}

__global__ void ScanPartialSums(float *PartialSums, int N)
{
#if DEBUG_PARTIAL_SUMS
    for (int I = 0; I < N; I++)
    {
        printf("PartialSums[%d] = %.3f\n", I, PartialSums[I]);
    }
#endif
    for (int I = 1; I < N; I++)
    {
        PartialSums[I] += PartialSums[I - 1];
    }
#if DEBUG_PARTIAL_SUMS
    for (int I = 0; I < N; I++)
    {
        printf("PartialSums[%d] = %.3f\n", I, PartialSums[I]);
    }
#endif
}

__global__ void ExpandPartialSums(const float *PartialSums, float *Output, int N)
{
    const int IsFirstBlock = blockIdx.x == 0;
    const int Tid = blockDim.x * blockIdx.x + threadIdx.x;
    const int TidInRange = Tid < N;

    __shared__ float BlockPartialSum;

    if (!IsFirstBlock)
    {
        if (threadIdx.x == 0)
        {
            BlockPartialSum = PartialSums[blockIdx.x - 1];
        }
        __syncthreads();

        if (TidInRange)
        {
            Output[Tid] += BlockPartialSum;
        }
    }
}

// input, output are device pointers
void solve(const float *input, float *output, int N)
{
    const int NumberOfBlocks = (N + BLOCK_DIM - 1) / BLOCK_DIM;

    float *PartialSums;
    hipMalloc(&PartialSums, NumberOfBlocks * sizeof(float));

    CalculatePartialSums<<<NumberOfBlocks, BLOCK_DIM>>>(input, output, PartialSums, N);
    ScanPartialSums<<<1, 1>>>(PartialSums, NumberOfBlocks);
    // Could optimize by removing first block.
    ExpandPartialSums<<<NumberOfBlocks, BLOCK_DIM>>>(PartialSums, output, N);

    hipFree(PartialSums);
}
