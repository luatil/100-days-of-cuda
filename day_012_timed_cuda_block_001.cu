#include <hip/hip_runtime.h>

typedef unsigned int u32;

struct cuda_profile_block
{
    char const *Label;
    u32 GlobalIndex;
    hipEvent_t Start;
    hipEvent_t Stop;

    cuda_profile_block(const char *Label, u32 GlobalIndex)
    {
        hipEventCreate(&Start);
        hipEventCreate(&Stop);
        hipEventRecord(Start);

        Label = Label;
        GlobalIndex = GlobalIndex;
    }

    ~cuda_profile_block(void)
    {
        hipEventRecord(Stop);
        hipEventSynchronize(Stop);

        f32 Milliseconds = 0;
        hipEventElapsedTime(&Milliseconds, Start, Stop);

        fprintf(stdout, "%s execution time: %f ms\n", Label, Milliseconds);
        hipEventDestroy(Start);
        hipEventDestroy(Stop);
    }
};

#define TimedCudaBlock(Name) cuda_profile_block Name

#define NameConcat2(A, B) A##B
#define NameConcat(A, B) NameConcat2(A, B)
#define TimeCudaBlock(Name) cuda_profile_block NameConcat(Block, __LINE__)(Name, __COUNTER__ + 1);
#define TimeCudaFunction TimeCudaBlock(__func__)
