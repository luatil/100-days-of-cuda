#include <hip/hip_runtime.h>

typedef unsigned int u32;

struct cuda_profile_block
{
    char const *Label;
    u32 GlobalIndex;
    hipEvent_t Start;
    hipEvent_t Stop;

    cuda_profile_block(const char *Label, u32 GlobalIndex)
    {
        hipEventCreate(&Start);
        hipEventCreate(&Stop);
        hipEventRecord(Start);

        Label = Label;
        GlobalIndex = GlobalIndex;
    }

    ~cuda_profile_block(void)
    {
        hipEventRecord(Stop);
        hipEventSynchronize(Stop);

        u32 Milliseconds = 0;
        hipEventElapsedTime(&Milliseconds, Start, Stop);

        fprintf(stdout, "%s execution time: %f ms\n", Label, Milliseconds);
        hipEventDestroy(Start);
        hipEventDestroy(Stop);
    }
};

#define TIMED_CUDA_BLOCK(Name) cuda_profile_block Name

#define NAME_CONCAT2(A, B) A##B
#define NAME_CONCAT(A, B) NameConcat2(A, B)
#define TIME_CUDA_BLOCK(Name) cuda_profile_block NameConcat(Block, __LINE__)(Name, __COUNTER__ + 1);
#define TIME_CUDA_FUNCTION TimeCudaBlock(__func__)
