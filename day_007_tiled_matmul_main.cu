#include "hip/hip_runtime.h"
/*
 * Day 07: Tiled MatMul Kernel
 *
 * Based on chapter 5 from PMPP.
 *
 */
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

typedef float f32;
typedef unsigned char u8;
typedef unsigned int u32;
typedef int s32;

#include "day_001_macros.h"
#include "day_007_tiled_matmul_kernel.cu"

static f32 Eps = 1e-6;

int main()
{
    u32 N = 256;
    u32 M = 256;
    u32 SizeInBytes = sizeof(f32) * N * M;

    f32 *HostA = AllocateCPU(f32, N * M);
    f32 *HostB = AllocateCPU(f32, N * M);
    f32 *HostC = AllocateCPU(f32, N * M);

    for (u32 I = 0; I < (N * M); I++)
    {
        HostA[I] = 1.0f;
        HostB[I] = 2.0f;
    }

    f32 *DeviceA, *DeviceB, *DeviceC;
    hipMalloc(&DeviceA, SizeInBytes);
    hipMalloc(&DeviceB, SizeInBytes);
    hipMalloc(&DeviceC, SizeInBytes);

    hipMemcpy(DeviceA, HostA, SizeInBytes, hipMemcpyHostToDevice);
    hipMemcpy(DeviceB, HostB, SizeInBytes, hipMemcpyHostToDevice);

    dim3 ThreadsPerBlock(16, 16, 1);
    dim3 BlocksPerGrid((N + 16 - 1) / 16, (M + 16 - 1) / 16, 1);

    TiledMatmulKernel<<<BlocksPerGrid, ThreadsPerBlock>>>(DeviceA, DeviceB, DeviceC, N, N, N);

    hipMemcpy(HostC, DeviceC, SizeInBytes, hipMemcpyDeviceToHost);

    for (u32 I = 0; I < (N * M); I++)
    {
        f32 Exp = 2.0f * 256.0f;
        f32 Diff = HostC[I] - Exp;
        if (abs(Diff) > Eps)
        {
            printf("Cuda Kernel Failed | Pos: %d | Expected %f Got %f", I, Exp, HostC[I]);
            exit(1);
        }
    }

    printf("DAY_07: CUDA SUCCESS");
}
