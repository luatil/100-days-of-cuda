#include <catch2/catch_all.hpp>
#include <random>
#include "day_076_vector_add.cuh"

TEST_CASE("VectorAdd property-based tests", "[cuda][vector_add][property]")
{
  // Property 1: Commutativity - A + B = B + A
  SECTION("Commutativity: A + B = B + A")
  {
    auto N = GENERATE(128, 256, 512, 1024, 2048);
    auto seed = GENERATE(take(5, random(0, 10000)));

    std::mt19937 rng(seed);
    std::uniform_real_distribution<float> dist(-100.0f, 100.0f);

    float *A = (float*)malloc(sizeof(float)*N);
    float *B = (float*)malloc(sizeof(float)*N);
    float *C1 = (float*)malloc(sizeof(float)*N);
    float *C2 = (float*)malloc(sizeof(float)*N);

    // Generate random inputs
    for (int I = 0; I < N; I++)
    {
      A[I] = dist(rng);
      B[I] = dist(rng);
    }

    size_t size = N * sizeof(float);
    float *d_A, *d_B, *d_C1, *d_C2;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C1, size);
    hipMalloc(&d_C2, size);

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    int BlockDim = 256;
    int GridDim = (N + BlockDim - 1) / BlockDim;

    // Compute A + B
    VectorAdd<<<GridDim, BlockDim>>>(d_A, d_B, d_C1, N);
    // Compute B + A
    VectorAdd<<<GridDim, BlockDim>>>(d_B, d_A, d_C2, N);

    hipMemcpy(C1, d_C1, size, hipMemcpyDeviceToHost);
    hipMemcpy(C2, d_C2, size, hipMemcpyDeviceToHost);

    // Verify commutativity
    for (int I = 0; I < N; I++)
    {
      REQUIRE(Abs(C1[I] - C2[I]) < 1e-5f);
    }

    hipFree(d_A); hipFree(d_B); hipFree(d_C1); hipFree(d_C2);
    free(A); free(B); free(C1); free(C2);
  }

  // Property 2: Identity - A + 0 = A
  SECTION("Identity: A + 0 = A")
  {
    auto N = GENERATE(128, 512, 1024);
    auto seed = GENERATE(take(3, random(0, 10000)));

    std::mt19937 rng(seed);
    std::uniform_real_distribution<float> dist(-100.0f, 100.0f);

    float *A = (float*)malloc(sizeof(float)*N);
    float *Zero = (float*)malloc(sizeof(float)*N);
    float *C = (float*)malloc(sizeof(float)*N);

    for (int I = 0; I < N; I++)
    {
      A[I] = dist(rng);
      Zero[I] = 0.0f;
    }

    size_t size = N * sizeof(float);
    float *d_A, *d_Zero, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_Zero, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_Zero, Zero, size, hipMemcpyHostToDevice);

    int BlockDim = 256;
    int GridDim = (N + BlockDim - 1) / BlockDim;
    VectorAdd<<<GridDim, BlockDim>>>(d_A, d_Zero, d_C, N);

    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    for (int I = 0; I < N; I++)
    {
      REQUIRE(Abs(C[I] - A[I]) < 1e-5f);
    }

    hipFree(d_A); hipFree(d_Zero); hipFree(d_C);
    free(A); free(Zero); free(C);
  }

  // Property 3: Associativity - (A + B) + C = A + (B + C)
  SECTION("Associativity: (A + B) + C = A + (B + C)")
  {
    auto N = GENERATE(256, 1024);
    auto seed = GENERATE(take(3, random(0, 10000)));

    std::mt19937 rng(seed);
    std::uniform_real_distribution<float> dist(-10.0f, 10.0f);

    float *A = (float*)malloc(sizeof(float)*N);
    float *B = (float*)malloc(sizeof(float)*N);
    float *C = (float*)malloc(sizeof(float)*N);
    float *Temp1 = (float*)malloc(sizeof(float)*N);
    float *Temp2 = (float*)malloc(sizeof(float)*N);
    float *Result1 = (float*)malloc(sizeof(float)*N);
    float *Result2 = (float*)malloc(sizeof(float)*N);

    for (int I = 0; I < N; I++)
    {
      A[I] = dist(rng);
      B[I] = dist(rng);
      C[I] = dist(rng);
    }

    size_t size = N * sizeof(float);
    float *d_A, *d_B, *d_C, *d_Temp1, *d_Temp2, *d_Result1, *d_Result2;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);
    hipMalloc(&d_Temp1, size);
    hipMalloc(&d_Temp2, size);
    hipMalloc(&d_Result1, size);
    hipMalloc(&d_Result2, size);

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
    hipMemcpy(d_C, C, size, hipMemcpyHostToDevice);

    int BlockDim = 256;
    int GridDim = (N + BlockDim - 1) / BlockDim;

    // Compute (A + B) + C
    VectorAdd<<<GridDim, BlockDim>>>(d_A, d_B, d_Temp1, N);
    VectorAdd<<<GridDim, BlockDim>>>(d_Temp1, d_C, d_Result1, N);

    // Compute A + (B + C)
    VectorAdd<<<GridDim, BlockDim>>>(d_B, d_C, d_Temp2, N);
    VectorAdd<<<GridDim, BlockDim>>>(d_A, d_Temp2, d_Result2, N);

    hipMemcpy(Result1, d_Result1, size, hipMemcpyDeviceToHost);
    hipMemcpy(Result2, d_Result2, size, hipMemcpyDeviceToHost);

    for (int I = 0; I < N; I++)
    {
      REQUIRE(Abs(Result1[I] - Result2[I]) < 1e-4f);
    }

    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    hipFree(d_Temp1); hipFree(d_Temp2);
    hipFree(d_Result1); hipFree(d_Result2);
    free(A); free(B); free(C);
    free(Temp1); free(Temp2);
    free(Result1); free(Result2);
  }
}
