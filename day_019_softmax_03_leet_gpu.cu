#include "hip/hip_runtime.h"
#include "day_019_common.h"
#include <cfloat>

typedef uint8_t u8;
typedef uint32_t u32;
typedef uint64_t u64;
typedef float f32;

#define MAX(_a, _b) (_a > _b) ? _a : _b
#define MIN(_a, _b) (_a < _b) ? _a : _b

#define BLOCK_DIM 256
#define COARSE_FACTOR 2
__device__ float AtomicMaxFloat(float *Address, float Val)
{
    int *AddressAsInt = (int *)Address;
    int Old = *AddressAsInt, Assumed;
    do
    {
        Assumed = Old;
        Old = atomicCAS(AddressAsInt, Assumed, __float_as_int(fmaxf(Val, __int_as_float(Assumed))));
    } while (Assumed != Old);
    return __int_as_float(Old);
}

__global__ void SoftMaxKernel02GlobalMax(const f32 *Input, f32 *GlobalMax, u32 N)
{
    __shared__ f32 Shared[BLOCK_DIM];

    u32 Segment = COARSE_FACTOR * blockDim.x * blockIdx.x;
    u32 Tid = Segment + threadIdx.x;
    u32 Tx = threadIdx.x;

    Shared[Tx] = -FLT_MIN;
    for (u32 I = 0; I < COARSE_FACTOR; I++)
    {
        if ((Tid + BLOCK_DIM * I) < N)
        {
            Shared[Tx] = MAX(Shared[Tx], Input[Tid + BLOCK_DIM * I]);
        }
    }

    // printf("Tid = %d | Tx = %d | Shared[Tx] = %.5f\n", Tid, Tx, Shared[Tx]);
    for (u32 Stride = blockDim.x / 2; Stride >= 1; Stride /= 2)
    {
        __syncthreads();
        if (Tx < Stride)
        {
            Shared[Tx] = MAX(Shared[Tx], Shared[Tx + Stride]);
        }
    }

    __syncthreads();
    // printf("Tid = %d | Tx = %d | Shared[Tx] = %.5f\n", Tid, Tx, Shared[Tx]);
    if (Tx == 0)
    {
        AtomicMaxFloat(GlobalMax, Shared[0]);
    }
}

__global__ void SoftMaxKernel02GlobalMaxSum(const f32 *Input, const f32 *GlobalMax, f32 *GlobalMaxSum, u32 N)
{
    __shared__ f32 Shared[BLOCK_DIM];

    u32 Segment = COARSE_FACTOR * blockDim.x * blockIdx.x;
    u32 Tid = Segment + threadIdx.x;
    u32 Tx = threadIdx.x;

    f32 Sum = 0.0f;
    for (u32 I = 0; I < COARSE_FACTOR; I++)
    {
        if ((Tid + BLOCK_DIM * I) < N)
        {
            Sum += expf(Input[Tid + BLOCK_DIM * I] - *GlobalMax);
        }
    }
    Shared[Tx] = Sum;

    //  printf("Tid = %d | Tx = %d | Shared[Tx] = %.5f\n", Tid, Tx, Shared[Tx]);

    for (u32 Stride = blockDim.x / 2; Stride >= 1; Stride /= 2)
    {
        __syncthreads();
        if (Tx < Stride)
        {
            Shared[Tx] += Shared[Tx + Stride];
        }
    }

    // NOTE(luatil): Shared[0] already has expfed value
    if (Tx == 0)
    {
        atomicAdd(GlobalMaxSum, Shared[0]);
    }
}

__global__ void SoftMaxKernel02Map(const f32 *Input, const f32 *GlobalMax, const f32 *GlobalMaxSum, f32 *Output,
                                      u32 N)
{
    u32 Tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (Tid < N)
    {
        Output[Tid] = expf(Input[Tid] - *GlobalMax) / *GlobalMaxSum;
    }
}

static void GpuSoftMax02(const f32 *DeviceInput, f32 *DeviceOutput, u32 N)
{
    u32 ThreadsPerBlock = MIN(BLOCK_DIM, N);
    u32 BlocksPerGrid = (N + (ThreadsPerBlock * COARSE_FACTOR) - 1) / (ThreadsPerBlock * COARSE_FACTOR);

    f32 *DeviceGlobalMax, *DeviceGlobalMaxSum;
    hipMalloc(&DeviceGlobalMax, sizeof(f32));
    hipMalloc(&DeviceGlobalMaxSum, sizeof(f32));

    SoftMaxKernel02GlobalMax<<<BlocksPerGrid, ThreadsPerBlock>>>(DeviceInput, DeviceGlobalMax, N);
    // DbgCudaF32(Device_GlobalMax);
    SoftMaxKernel02GlobalMaxSum<<<BlocksPerGrid, ThreadsPerBlock>>>(DeviceInput, DeviceGlobalMax,
                                                                       DeviceGlobalMaxSum, N);

    // DbgCudaF32(Device_GlobalMaxSum);
    BlocksPerGrid = (N + (ThreadsPerBlock * 1) - 1) / (ThreadsPerBlock * 1);
    SoftMaxKernel02Map<<<BlocksPerGrid, ThreadsPerBlock>>>(DeviceInput, DeviceGlobalMax, DeviceGlobalMaxSum,
                                                              DeviceOutput, N);
    hipFree(DeviceGlobalMax);
    hipFree(DeviceGlobalMax);
}
#undef COARSE_FACTOR
#undef BLOCK_DIM

#ifdef LEET_GPU
#include "solve.h"
#include <cfloat>
#include <hip/hip_runtime.h>
#include <stdint.h>

typedef uint8_t u8;
typedef uint32_t u32;
typedef uint64_t u64;
typedef float f32;

#define Max(_a, _b) (_a > _b) ? _a : _b
#define Min(_a, _b) (_a < _b) ? _a : _b

#define BLOCK_DIM 256
#define COARSE_FACTOR 2
__device__ float atomicMaxFloat(float *address, float val)
{
    int *address_as_int = (int *)address;
    int old = *address_as_int, assumed;
    do
    {
        assumed = old;
        old = atomicCAS(address_as_int, assumed, __float_as_int(fmaxf(val, __int_as_float(assumed))));
    } while (assumed != old);
    return __int_as_float(old);
}

__global__ void SoftMax_Kernel_02_GlobalMax(const f32 *Input, f32 *GlobalMax, u32 N)
{
    __shared__ f32 Shared[BLOCK_DIM];

    u32 Segment = COARSE_FACTOR * blockDim.x * blockIdx.x;
    u32 Tid = Segment + threadIdx.x;
    u32 Tx = threadIdx.x;

    Shared[Tx] = -FLT_MIN;
    for (u32 I = 0; I < COARSE_FACTOR; I++)
    {
        if ((Tid + BLOCK_DIM * I) < N)
        {
            Shared[Tx] = Max(Shared[Tx], Input[Tid + BLOCK_DIM * I]);
        }
    }

    printf("Tid = %d | Tx = %d | Shared[Tx] = %.5f\n", Tid, Tx, Shared[Tx]);
    for (u32 Stride = (blockDim.x + 2 - 1) / 2; Stride >= 1; Stride /= 2)
    {
        __syncthreads();
        if (Tx < Stride)
        {
            Shared[Tx] = Max(Shared[Tx], Shared[Tx + Stride]);
        }
    }

    __syncthreads();
    if (Tx == 0)
    {
        printf("MAX: Tid = %d | Tx = %d | Shared[Tx] = %.5f\n", Tid, Tx, Shared[Tx]);
        atomicMaxFloat(GlobalMax, Shared[0]);
    }
}

__global__ void SoftMax_Kernel_02_GlobalMaxSum(const f32 *Input, const f32 *GlobalMax, f32 *GlobalMaxSum, u32 N)
{
    __shared__ f32 Shared[BLOCK_DIM];

    u32 Segment = COARSE_FACTOR * blockDim.x * blockIdx.x;
    u32 Tid = Segment + threadIdx.x;
    u32 Tx = threadIdx.x;

    f32 Sum = 0.0f;
    for (u32 I = 0; I < COARSE_FACTOR; I++)
    {
        if ((Tid + BLOCK_DIM * I) < N)
        {
            Sum += expf(Input[Tid + BLOCK_DIM * I] - *GlobalMax);
        }
    }
    Shared[Tx] = Sum;

    printf("Tid = %d | Tx = %d | Shared[Tx] = %.5f\n", Tid, Tx, Shared[Tx]);

    // for (u32 Stride = blockDim.x / 2; Stride >= 1; Stride /= 2)
    for (u32 Stride = (blockDim.x + 2 - 1) / 2; Stride >= 1; Stride /= 2)
    {
        __syncthreads();
        if (Tx < Stride)
        {
            Shared[Tx] += Shared[Tx + Stride];
        }
    }

    // NOTE(luatil): Shared[0] already has expfed value
    if (Tx == 0)
    {
        printf("SUM: Tid = %d | Tx = %d | Shared[Tx] = %.5f\n", Tid, Tx, Shared[Tx]);
        atomicAdd(GlobalMaxSum, Shared[0]);
    }
}

__global__ void SoftMax_Kernel_02_Map(const f32 *Input, const f32 *GlobalMax, const f32 *GlobalMaxSum, f32 *Output,
                                      u32 N)
{
    u32 Tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (Tid < N)
    {
        Output[Tid] = expf(Input[Tid] - *GlobalMax) / *GlobalMaxSum;
    }
}

// input, output are device pointers (i.e. pointers to memory on the GPU)
void solve(const float *input, float *output, int N)
{
    u32 ThreadsPerBlock = Min(BLOCK_DIM, N);
    u32 BlocksPerGrid = (N + (ThreadsPerBlock * COARSE_FACTOR) - 1) / (ThreadsPerBlock * COARSE_FACTOR);

    f32 *Device_GlobalMax, *Device_GlobalMaxSum;
    hipMalloc(&Device_GlobalMax, sizeof(f32));
    hipMalloc(&Device_GlobalMaxSum, sizeof(f32));

    SoftMax_Kernel_02_GlobalMax<<<BlocksPerGrid, ThreadsPerBlock>>>(input, Device_GlobalMax, N);
    SoftMax_Kernel_02_GlobalMaxSum<<<BlocksPerGrid, ThreadsPerBlock>>>(input, Device_GlobalMax, Device_GlobalMaxSum, N);

    // DbgCudaF32(Device_GlobalMaxSum);
    BlocksPerGrid = (N + (ThreadsPerBlock * 1) - 1) / (ThreadsPerBlock * 1);
    SoftMax_Kernel_02_Map<<<BlocksPerGrid, ThreadsPerBlock>>>(input, Device_GlobalMax, Device_GlobalMaxSum, output, N);
    hipFree(Device_GlobalMax);
    hipFree(Device_GlobalMax);
}

#endif
