#include "hip/hip_runtime.h"
#include "day_015_common.h"

#define BLOCK_DIM 256

__global__ void ReductionKernel(f32 *Input, f32 *Output, u32 N)
{
    __shared__ f32 Shared[BLOCK_DIM];

    u32 Segment = 2 * blockDim.x * blockIdx.x;
    u32 Tid = Segment + threadIdx.x;
    u32 Tx = threadIdx.x;

    Shared[Tx] = Input[Tid] + Input[Tid + BLOCK_DIM];

    for (u32 Stride = blockDim.x / 2; Stride >= 1; Stride /= 2)
    {
        __syncthreads();
        if (Tx < Stride)
        {
            Shared[Tx] += Shared[Tx + Stride];
        }
    }

    if (Tx == 0)
    {
        atomicAdd(Output, Shared[0]);
    }
}

int main()
{
    const u32 N = BLOCK_DIM * 256;
    f32 *Input = AllocateCPU(f32, N);

    for (u32 I = 0; I < N; I++)
    {
        Input[I] = 1.0f;
    }

    f32 *DeviceInput, *DeviceOutput;
    hipMalloc(&DeviceInput, sizeof(f32) * N);
    hipMalloc(&DeviceOutput, sizeof(f32) * 1);
    hipMemset(DeviceOutput, 0, sizeof(f32));

    hipMemcpy(DeviceInput, Input, sizeof(f32) * N, hipMemcpyHostToDevice);

    u32 ThreadsPerBlock = BLOCK_DIM;
    u32 BlocksPerGrid = (N + (BLOCK_DIM * 2) - 1) / (BLOCK_DIM * 2);

    ReductionKernel<<<BlocksPerGrid, ThreadsPerBlock>>>(DeviceInput, DeviceOutput, N);

    f32 Output;
    hipMemcpy(&Output, DeviceOutput, sizeof(f32), hipMemcpyDeviceToHost);

    fprintf(stdout, "%f\n", Output);
    Assert((N - Output) < 0.1);
}
