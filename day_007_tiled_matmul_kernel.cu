/*
 * Day 07: Tiled MatMul Kernel
 *
 * Based on chapter 5 from PMPP.
 *
 */
#include <hip/hip_runtime.h>
#include <stdlib.h>

typedef float f32;
typedef unsigned char u8;
typedef unsigned int u32;
typedef int s32;

#ifndef TILE_WIDTH
#define TILE_WIDTH 16
#endif

__global__ void TiledMatmulKernel(f32 *InputA, f32 *InputB, f32 *Output, u32 HeightA, u32 WidthA, u32 WidthB)
{
    __shared__ f32 TileA[TILE_WIDTH][TILE_WIDTH];
    __shared__ f32 TileB[TILE_WIDTH][TILE_WIDTH];

    u32 Row = blockDim.y * blockIdx.y + threadIdx.y;
    u32 Col = blockDim.x * blockIdx.x + threadIdx.x;

    u32 Tx = threadIdx.x;
    u32 Ty = threadIdx.y;

    f32 DotProduct = 0.0f;

    for (u32 I = 0; I < (WidthA + TILE_WIDTH - 1) / TILE_WIDTH; I++)
    {

        u32 IndexA = Row * WidthA + I * TILE_WIDTH + Tx;
        u32 IndexB = Col + Ty * WidthB + I * TILE_WIDTH * WidthB;

        TileA[Tx][Ty] = (IndexA < (HeightA * WidthA)) ? InputA[IndexA] : 0.0f;
        TileB[Tx][Ty] = (IndexB < (WidthA * WidthB)) ? InputB[IndexB] : 0.0f;

        __syncthreads();

        for (u32 K = 0; K < TILE_WIDTH; K++)
        {
            DotProduct += TileA[Tx][K] * TileB[K][Ty];
        }

        __syncthreads();
    }

    if (Row < HeightA && Col < WidthB)
    {
        Output[Row * WidthB + Col] = DotProduct;
    }
}
