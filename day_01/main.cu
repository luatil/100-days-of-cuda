
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

typedef float f32;
typedef unsigned int u32;

#define AllocateCPU(_Type, _NumberOfElements)                                  \
  (_Type *)malloc(sizeof(_Type) * _NumberOfElements)

#ifndef DEBUG_ENABLED
#define DEBUG_ENABLED 0
#endif

#if DEBUG_ENABLED
#define DbgU32(_Val) printf(#_Val "=%d\n", (_Val))
#define DbgF32(_Val) printf(#_Val "=%f\n", (_Val))
#else
#define DbgU32(_Val)
#define DbgF32(_Val)
#endif

static f32 Eps = 1e-6;

__global__ void AddKernel(f32 *InputA, f32 *InputB, f32 *Output, u32 Length) {
  u32 Tid = blockDim.x * blockIdx.x + threadIdx.x;
  if (Tid < Length) {
    Output[Tid] = InputA[Tid] + InputB[Tid];
  }
}

int main() {
  u32 N = 4096;
  u32 SizeInBytes = sizeof(f32) * N;

  f32 *HostA = AllocateCPU(f32, N);
  f32 *HostB = AllocateCPU(f32, N);
  f32 *HostC = AllocateCPU(f32, N);

  for (u32 I = 0; I < N; I++) {
    HostA[I] = 1.0f;
    HostB[I] = 2.0f;
  }

  f32 *DeviceA, *DeviceB, *DeviceC;
  hipMalloc(&DeviceA, SizeInBytes);
  hipMalloc(&DeviceB, SizeInBytes);
  hipMalloc(&DeviceC, SizeInBytes);

  hipMemcpy(DeviceA, HostA, SizeInBytes, hipMemcpyHostToDevice);
  hipMemcpy(DeviceB, HostB, SizeInBytes, hipMemcpyHostToDevice);

  u32 NumberOfThreads = 32;
  u32 NumberOfBlocks = (N + NumberOfThreads - 1) / NumberOfThreads;

  DbgU32(NumberOfThreads);
  DbgU32(NumberOfBlocks);
  DbgU32(NumberOfThreads * NumberOfBlocks);

  dim3 Dim3NumberOfThreads(NumberOfThreads);
  dim3 Dim3NumberOfBlocks(NumberOfBlocks);

  AddKernel<<<Dim3NumberOfBlocks, Dim3NumberOfThreads>>>(DeviceA, DeviceB,
                                                         DeviceC, N);

  hipMemcpy(HostC, DeviceC, SizeInBytes, hipMemcpyDeviceToHost);

  for (u32 I = 0; I < N; I++) {
    f32 Diff = HostC[I] - 3.0f;
    if (abs(Diff) > Eps) {
      printf("Cuda Kernel Failed | Pos: %d | Expected %f Got %f", I, 3.0f,
             HostC[I]);
      exit(1);
    }
  }

  printf("DAY_01: CUDA SUCCESS");
}
