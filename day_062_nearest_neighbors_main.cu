#include <hip/hip_runtime.h>
#include <cfloat>
#include <stdio.h>
// #define LEET_GPU

// Ideas to improve this:
// In some way or another we need to use some sort of memory technique
// to speed up the process.
//
// One of them could be to use a grid and only look at elements that fall
// occur withing that grid.
//
// But this makes assumptions on what the grid is actually like.

__global__ void NearestNeighbors(const float *Points, int *Indices, int N)
{
    int PointIdx = blockIdx.x * blockDim.x + threadIdx.x;

    if (PointIdx < N)
    {
        // Iterate over all of the other points
        // Check which one is closer to our current point
        float MinDistanceSquared = FLT_MAX;
        for (int I = 0; I < N; I++)
        {
            if (PointIdx != I)
            {
                float Distance = powf(Points[3 * PointIdx] - Points[3 * I], 2) +
                                 powf(Points[3 * PointIdx + 1] - Points[3 * I + 1], 2) +
                                 powf(Points[3 * PointIdx + 2] - Points[3 * I + 2], 2);
                if (Distance < MinDistanceSquared)
                {
                    Indices[PointIdx] = I;
                    MinDistanceSquared = Distance;
                }
            }
        }
    }
}

extern "C" void solve(const float* points, int* indices, int N)
{
    const int BlockDim = 256;
    const int GridDim = (N + BlockDim - 1) / BlockDim;
    NearestNeighbors<<<GridDim, BlockDim>>>(points, indices, N);
}

#ifndef LEET_GPU
int main()
{
    // points  = [(0,0,0), (1,0,0), (5,5,5)]
    //         indices = [-1, -1, -1]
    //         N       = 3
    float Points[] = {0.0f, 0.0f, 0.0f, 1.0f, 0.0f, 0.0f, 5.0f, 5.0f, 5.0f};
    int Indices[] = {-1, -1, -1};
    int N = 3;

    float *d_Points;
    int  *d_Indices;
    hipMalloc(&d_Points, N * 3 * sizeof(float));
    hipMalloc(&d_Indices, N * sizeof(int));

    hipMemcpy(d_Points, Points, N * 3 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Indices, Indices, N * sizeof(int), hipMemcpyHostToDevice);

    solve(d_Points, d_Indices, N);

    hipMemcpy(Indices, d_Indices, N * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < N; i++)
    {
        printf("%d => %d\n", i, Indices[i]);
    }
}
#endif
