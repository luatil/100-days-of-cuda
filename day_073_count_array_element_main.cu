
#include <hip/hip_runtime.h>
#include <stdio.h>

#define ArraySize(_X) (sizeof(_X) / sizeof(_X[0]))

__global__ void CountEqualKernel(const int *Input, int *Output, int N, int K)
{
    int Tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (Tid < N && Input[Tid] == K)
    {
        atomicAdd(Output, 1);
    }
}

int main()
{
  int Input[] = {1, 2, 3, 4, 1};
  int K = 1;

  int *DeviceInput;
  hipMalloc(&DeviceInput, sizeof(int) * 5);
  hipMemcpy(DeviceInput, Input, sizeof(int) * 5, hipMemcpyHostToDevice);

  int *DeviceOutput;
  hipMalloc(&DeviceOutput, sizeof(int));
  hipMemset(DeviceOutput, 0, sizeof(int));

  int BlockDim = 256;
  int GridDim = ArraySize(Input);

  CountEqualKernel<<<GridDim, BlockDim>>>(DeviceInput, DeviceOutput, 5, K);

  hipDeviceSynchronize();

  int Result;
  hipMemcpy(&Result, DeviceOutput, sizeof(int), hipMemcpyDeviceToHost);

  printf("Number of elements equal to %d: %d\n", K, Result);
}
