/*
 * Day 008: Generate Random Matrix
 */
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <stdint.h>
#include <stdlib.h>

typedef float f32;
typedef uint32_t u32;
typedef uint64_t u64;

typedef int32_t s32;

__global__ void GenerateSequentialMatrix(f32 *OutputMatrix, u32 Width, u32 Height)
{
    u32 Tid = blockIdx.x * blockDim.x + threadIdx.x;
    u32 TotalElements = Width * Height;

    if (Tid < TotalElements)
    {
        OutputMatrix[Tid] = (f32)Tid;
    }
}
