#include "hip/hip_runtime.h"
#include "day_015_common.h"

#define BLOCK_DIM 256
#define COARSE_FACTOR 4 // Min Is 2

__global__ void ReductionKernel(f32 *Input, f32 *Output, u32 N)
{
    __shared__ f32 Shared[BLOCK_DIM];

    u32 Segment = COARSE_FACTOR * blockDim.x * blockIdx.x;
    u32 Tid = Segment + threadIdx.x;
    u32 Tx = threadIdx.x;

    for (u32 I = 0; I < COARSE_FACTOR; I++)
    {
        Shared[Tx] += Input[Tid + BLOCK_DIM * I];
    }

    for (u32 Stride = blockDim.x / 2; Stride >= 1; Stride /= 2)
    {
        __syncthreads();
        if (Tx < Stride)
        {
            Shared[Tx] += Shared[Tx + Stride];
        }
    }

    if (Tx == 0)
    {
        atomicAdd(Output, Shared[0]);
    }
}

int main()
{
    const u32 N = BLOCK_DIM * 256;
    f32 *Input = AllocateCPU(f32, N);

    for (u32 I = 0; I < N; I++)
    {
        Input[I] = 1.0f;
    }

    f32 *Device_Input, *Device_Output;
    hipMalloc(&Device_Input, sizeof(f32) * N);
    hipMalloc(&Device_Output, sizeof(f32) * 1);
    hipMemset(Device_Output, 0, sizeof(f32));

    hipMemcpy(Device_Input, Input, sizeof(f32) * N, hipMemcpyHostToDevice);

    u32 ThreadsPerBlock = BLOCK_DIM;
    u32 BlocksPerGrid = (N + (BLOCK_DIM * COARSE_FACTOR) - 1) / (BLOCK_DIM * COARSE_FACTOR);

    ReductionKernel<<<BlocksPerGrid, ThreadsPerBlock>>>(Device_Input, Device_Output, N);

    f32 Output;
    hipMemcpy(&Output, Device_Output, sizeof(f32), hipMemcpyDeviceToHost);

    fprintf(stdout, "Expected: %f\n", N * 1.0f);
    fprintf(stdout, "Output: %f\n", Output);
    Assert((N - Output) < 0.1);
}
