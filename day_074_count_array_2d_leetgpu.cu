#include <hip/hip_runtime.h>

__global__ void count_2d_equal_kernel(const int *input, int *output, int N,
                                      int M, int K) {
  int Row = blockDim.x * blockIdx.x + threadIdx.x;
  int Col = blockDim.y * blockIdx.y + threadIdx.y;

  if (Row < N && Col < M) {
    int Pos = Row * M + Col;
    if (input[Pos] == K) {
      atomicAdd(output, 1);
    }
  }
}

// input, output are device pointers (i.e. pointers to memory on the GPU)
extern "C" void solve(const int *input, int *output, int N, int M, int K) {
  dim3 threadsPerBlock(16, 16);
  dim3 blocksPerGrid((M + threadsPerBlock.x - 1) / threadsPerBlock.x,
                     (N + threadsPerBlock.y - 1) / threadsPerBlock.y);

  count_2d_equal_kernel<<<blocksPerGrid, threadsPerBlock>>>(input, output, N, M,
                                                            K);
  hipDeviceSynchronize();
}
