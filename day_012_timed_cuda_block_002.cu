#include "day_001_macros.h"
#include <hip/hip_runtime.h>
#include <stdint.h>

typedef uint32_t u32;
typedef uint64_t u64;
typedef float f32;
typedef double f64;

struct cuda_profile_block
{
    char const *Label;
    u32 GlobalIndex;
    u64 BytesRead;
    u64 BytesWritten;
    f32 FlopsPerByte;
    hipEvent_t Start;
    hipEvent_t Stop;

    cuda_profile_block(const char *Label, u32 GlobalIndex, u64 BytesRead, u64 BytesWritten, f32 FlopsPerByte)
    {
        hipEventCreate(&Start);
        hipEventCreate(&Stop);
        hipEventRecord(Start);

        Label = Label;
        GlobalIndex = GlobalIndex;
        BytesRead = BytesRead;
        BytesWritten = BytesWritten;
        FlopsPerByte = FlopsPerByte;
    }

    ~cuda_profile_block(void)
    {
        hipEventRecord(Stop);
        hipEventSynchronize(Stop);

        f32 Milliseconds = 0;
        hipEventElapsedTime(&Milliseconds, Start, Stop);

        f32 Megabyte = 1024.0f * 1024.0f;
        f32 Gigabyte = Megabyte * 1024.0f;

        u64 BytesProcessed = BytesRead + BytesWritten;
        f32 TotalFlops = FlopsPerByte * BytesProcessed;
        f32 FlopsPerSecond = (TotalFlops / Milliseconds) * 1000.0f;

        f32 Bandwidth = (BytesProcessed / Milliseconds) * 1000.0f;

        fprintf(stdout, "%s | Execution Time: %f ms\n", Label, Milliseconds);

        if (BytesProcessed && FlopsPerByte)
        {
            fprintf(stdout, "%s | Bytes Processed: %.2f Mb\n", Label, BytesProcessed / Megabyte);
            fprintf(stdout, "%s | Effective Bandwidth: %.4f Gb/s\n", Label, Bandwidth / Gigabyte);
            fprintf(stdout, "%s | Compute Throughput: %.4f GFLOPS/s\n", Label, FlopsPerSecond / Gigabyte);
        }

        fprintf(stdout, "-------------------------\n");

        hipEventDestroy(Start);
        hipEventDestroy(Stop);
    }
};

#define NAME_CONCAT2(A, B) A##B
#define NAME_CONCAT(A, B) NameConcat2(A, B)
#define TIME_CUDA_BLOCK(Name) cuda_profile_block NameConcat(Block, __LINE__)(Name, __COUNTER__ + 1, 0, 0, 0.0f);
#define TIME_CUDA_BANDWIDTH(Name, BytesRead_, BytesWritten_, FlopsPerByte_)                                              \
    cuda_profile_block NameConcat(Block, __LINE__)(Name, __COUNTER__ + 1, BytesRead_, BytesWritten_, FlopsPerByte_);
#define TIME_CUDA_FUNCTION TimeCudaBlock(__func__)
