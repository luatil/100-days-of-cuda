#include <stdio.h>

#include "day_001_macros.h"
#include "day_001_vector_add_kernel.cu"

int main(int ArgumentCount, char **Arguments)
{
    u32 ExitCode = 0;

    if (ArgumentCount == 2)
    {
        u32 N = 0;
        sscanf(Arguments[1], "%d", &N);

        u32 SizeInBytes = sizeof(f32) * N;

        f32 *HostA = AllocateCPU(f32, N);
        f32 *HostB = AllocateCPU(f32, N);
        f32 *HostC = AllocateCPU(f32, N);

        for (u32 I = 0; I < N; I++)
        {
            HostA[I] = 1.0f * I;
            HostB[I] = 2.0f * I;
        }

        f32 *DeviceA, *DeviceB, *DeviceC;

        // Allocations
        {
            hipMalloc(&DeviceA, SizeInBytes);
            hipMalloc(&DeviceB, SizeInBytes);
            hipMalloc(&DeviceC, SizeInBytes);
        }

        // Memcpy Host To Device
        {
            hipMemcpy(DeviceA, HostA, SizeInBytes, hipMemcpyHostToDevice);
            hipMemcpy(DeviceB, HostB, SizeInBytes, hipMemcpyHostToDevice);
        }

        // Kernel Launch
        {
            u32 ThreadsPerBlock = 32;
            u32 BlocksPerGrid = (N + ThreadsPerBlock - 1) / ThreadsPerBlock;

            hipEvent_t Start, Stop;
            hipEventCreate(&Start);
            hipEventCreate(&Stop);

            hipEventRecord(Start);

            AddKernel<<<BlocksPerGrid, ThreadsPerBlock>>>(DeviceA, DeviceB, DeviceC, N);

            hipEventRecord(Stop);
            hipEventSynchronize(Stop);

            f32 Milliseconds = 0;
            hipEventElapsedTime(&Milliseconds, Start, Stop);

            fprintf(stdout, "Kernel execution time: %f ms\n", Milliseconds);
            hipEventDestroy(Start);
            hipEventDestroy(Stop);
        }

        // Memcpy Device To Host
        {
            hipMemcpy(HostC, DeviceC, SizeInBytes, hipMemcpyDeviceToHost);
        }

        for (u32 I = 0; I < N; I++)
        {
            printf("%.1f ", HostC[I]);
        }
        printf("\n");
    }
    else
    {
        fprintf(stderr, "Usage: %s [number of vector elements]\n", Arguments[0]);
        ExitCode = 1;
    }

    return ExitCode;
}
