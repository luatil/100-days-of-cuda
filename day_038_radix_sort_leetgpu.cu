
#include <hip/hip_runtime.h>
// NOTE(luatil): Second solution to RadixSort still does not work
#define LEET_GPU_NO_IMPORT
#ifndef LEET_GPU_NO_IMPORT
#include "solve.h"
#include <cuda_runtime.h>
#endif

#define SOLUTION 1

#define BLOCK_SIZE 256
#define RADIX_BITS 4
#define RADIX_SIZE (1 << RADIX_BITS) // 16 for 4-bit radix

__device__ void Swap(unsigned int *A, int I, int J)
{
    unsigned int Temp = A[I];
    A[I] = A[J];
    A[J] = Temp;
}

#if SOLUTION == 0
__global__ void RadixSort(const unsigned int *input, unsigned int *output, int N)
{
    for (int i = 0; i < N; i++)
    {
        output[i] = input[i];
    }

    // NOTE(luatil): I can't believe it sorts algorithm:
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            if (output[i] < output[j])
            {
                Swap(output, i, j);
            }
        }
    }
}
#elif SOLUTION == 1
__global__ void RadixSort(const unsigned int *Input, unsigned int *Output, int N)
{
    int Tid = blockDim.x * blockIdx.x + threadIdx.x;
    int Tx = threadIdx.x;

    if (Tid < N)
    {
        Output[Tid] = Input[Tid];
    }
    __syncthreads();

    for (int Bit = 0; Bit < 32; Bit++)
    {
        __shared__ unsigned int Shared[BLOCK_SIZE];
        __shared__ int Zeros;
        __shared__ int Ones;

        if (Tx == 0)
        {
            Zeros = 0;
            Ones = 0;
        }
        __syncthreads();

        if (Tid < N)
        {
            Shared[Tx] = Output[Tid];
        }
        __syncthreads();

        if (Tid < N)
        {
            int BitVal = (Shared[Tx] >> Bit) & 1;
            if (BitVal == 0)
            {
                atomicAdd(&Zeros, 1);
            }
            else
            {
                atomicAdd(&Ones, 1);
            }
        }
        __syncthreads();

        // Partition based on bit value
        if (Tid < N)
        {
            int BitVal = (Shared[Tx] >> Bit) & 1;
            int Pos = 0;

            if (BitVal == 0)
            {
                // Count zeros before this position
                for (int I = 0; I < Tx; I++)
                {
                    if (Tid - Tx + I < N)
                    {
                        int PrevBit = (Shared[I] >> Bit) & 1;
                        if (PrevBit == 0)
                            Pos++;
                    }
                }
            }
            else
            {
                // Count ones before this position
                Pos = Ones;
                for (int I = 0; I < Tx; I++)
                {
                    if (Tid - Tx + I < N)
                    {
                        int PrevBit = (Shared[I] >> Bit) & 1;
                        if (PrevBit == 1)
                            Pos++;
                    }
                }
            }

            if (Pos < N)
            {
                Output[blockIdx.x * blockDim.x + Pos] = Shared[Tx];
            }
        }
        __syncthreads();
    }
}
#endif

// input, output are device pointers
void Solve(const unsigned int *Input, unsigned int *Output, int N)
{
    int GridDim = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    RadixSort<<<GridDim, BLOCK_SIZE>>>(Input, Output, N);
}
