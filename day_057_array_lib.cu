
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void AddKernel(float *A, float *B, float *C, int N)
{
    int Tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (Tid < N)
    {
        C[Tid] = A[Tid] + B[Tid];
    }
}

__global__ void MultiplyKernel(float *A, float *B, float *C, int N)
{
    int Tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (Tid < N)
    {
        C[Tid] = A[Tid] * B[Tid];
    }
}

__global__ void MultiplyConstantKernel(float *A, float *B, float X, int N)
{
    int Tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (Tid < N)
    {
        B[Tid] = A[Tid] * X;
    }
}

__global__ void LinSpace(float *X, float Start, float End, int N)
{
    int Tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (Tid < N)
    {
        X[Tid] = Start + Tid * ((End - Start) / (N - 1));
    }
}

template <int BlockDim = 256> __global__ void SumKernel(float *X, float *Result, int N)
{
    __shared__ float Shared[BlockDim];

    int Tid = blockIdx.x * blockDim.x + threadIdx.x;
    int Tx = threadIdx.x;

    Shared[Tx] = Tid < N ? X[Tid] : 0.0f;
    __syncthreads();

    for (int Stride = BlockDim / 2; Stride > 0; Stride /= 2)
    {
        if (Tx < Stride)
        {
            Shared[Tx] += Shared[Tx + Stride];
        }
        __syncthreads();
    }

    if (Tx == 0)
    {
        atomicAdd(Result, Shared[0]);
    }
}

template <int BlockDim = 256> __global__ void CopyKernel(float *V, float X, int N)
{
    int Tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (Tid < N)
    {
        V[Tid] = X;
    }
}

__global__ void PrintKernel(float *X, int N)
{
    for (int I = 0; I < N; I++)
    {
        printf("%f\n", X[I]);
    }
}

__device__ __host__ int Ceil(int Num, int Den)
{
    return (Num + Den - 1) / Den;
}

template <int Dim> struct array
{
    float *MData;

    array(float X)
    {
        hipMalloc(&MData, sizeof(float) * Dim);
        CopyKernel<<<Ceil(Dim, 256), 256>>>(MData, X, Dim);
    }

    array(array &&Other) noexcept : MData(Other.MData)
    {
        Other.MData = nullptr;
    }

    array &operator=(array &&Other) noexcept
    {
        if (this != &Other)
        {
            hipFree(MData);
            MData = Other.MData;
            Other.MData = nullptr;
        }
        return *this;
    }

    array(const array &) = delete;
    array &operator=(const array &) = delete;

    ~array()
    {
        if (MData)
        {
            hipFree(MData);
        }
    }
};

template <int Dim> array<Dim> operator+(const array<Dim> &A, const array<Dim> &B)
{
    array<Dim> Result(0.0f);
    AddKernel<<<Ceil(Dim, 256), 256>>>(A.MData, B.MData, Result.MData, Dim);
    return Result;
}

template <int Dim> array<Dim> operator*(const array<Dim> &A, const array<Dim> &B)
{
    array<Dim> Result(0.0f);
    MultiplyKernel<<<Ceil(Dim, 256), 256>>>(A.MData, B.MData, Result.MData, Dim);
    return Result;
}

template <int Dim> array<Dim> operator*(const array<Dim> &A, float X)
{
    array<Dim> Result(0.0f);
    MultiplyConstantKernel<<<Ceil(Dim, 256), 256>>>(A.MData, Result.MData, X, Dim);
    return Result;
}

template <int Dim> array<1> Sum(const array<Dim> &A)
{
    array<1> Result(0.0f);
    SumKernel<<<Ceil(Dim, 256), 256>>>(A.MData, Result.MData, Dim);
    return Result;
}

template <int Dim> void Print(const array<Dim> &Array)
{
    PrintKernel<<<1, 1>>>(Array.MData, Dim);
}

int main()
{
    const int N = 1024;

    array<N> Ones(1.0f);
    array<N> Twos(2.0f);
    array<N> Threes = Ones + Twos;
    array<N> Six = Threes * Twos;
    array<N> Twelve = Six * 2.0f;
    array<1> TwelveSum = Sum(Twelve);

    Print(TwelveSum);
    hipDeviceSynchronize();
}
