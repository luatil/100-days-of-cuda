/**
 * Computes the softmax of two vectors A and B.
 *
 *
 * Input:  A = [1.0, 2.0, 3.0, 4.0]
 * B = [5.0, 6.0, 7.0, 8.0]
 * Output: result = 70.0  (1.0*5.0 + 2.0*6.0 + 3.0*7.0 + 4.0*8.0)
 *
 *
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#define SOLUTION_00 0
#define SOLUTION_01 0
#define SOLUTION_02 1

#if SOLUTION_00
// Naive version with just <<<1,1>>> launch
__global__ void Kernel_DotProduct(const float *A, const float *B, float *result, int N)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    float Sum = 0.0f;
    for (int i = 0; i < N; i++)
    {
        Sum += A[i] * B[i];
    }
    *result = Sum;
}
// A, B, result are device pointers
void solve(const float *A, const float *B, float *result, int N)
{
    Kernel_DotProduct_00<<<1, 1>>>(A, B, result, N);
}
#elif SOLUTION_01
#define BLOCK_DIM 1024
// Version with <<<1, 1024>>>
__global__ void Kernel_DotProduct(const float *A, const float *B, float *result, int N)
{
    __shared__ float SharedMem[BLOCK_DIM];

    int tx = threadIdx.x;

    // ThreadCoarsening
    float Sum = 0.0f;
    for (int i = 0; i < ((N + blockDim.x - 1) / blockDim.x); i++)
    {
        int pos = blockDim.x * i + tx;
        if (pos < N)
        {
            Sum += A[pos] * B[pos];
        }
    }
    SharedMem[tx] = Sum;
    __syncthreads();

    for (int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        __syncthreads();
        if (tx < s)
        {
            SharedMem[tx] += SharedMem[tx + s];
        }
    }
    __syncthreads();

    if (tx == 0)
    {
        *result = SharedMem[0];
    }
}

#define BLOCK_DIM 512
#define COARSE_FACTOR 4

__global__ void KernelDotProduct02(const float *A, const float *B, float *Result, int N)
{
    __shared__ float SharedMem[BLOCK_DIM];

    int Tid = COARSE_FACTOR * blockDim.x * blockIdx.x + threadIdx.x;
    int Tx = threadIdx.x;

    // ThreadCoarsening
    float Sum = 0.0f;
    for (int I = 0; I < COARSE_FACTOR; I++)
    {
        int Pos = Tid + blockDim.x * I;
        if (Pos < N)
        {
            Sum += A[Pos] * B[Pos];
        }
    }
    SharedMem[Tx] = Sum;
    __syncthreads();

    for (int S = blockDim.x / 2; S > 0; S >>= 1)
    {

        if (Tx < S)
        {
            SharedMem[Tx] += SharedMem[Tx + S];
        }
        __syncthreads();
    }

    if (Tx == 0)
    {
        atomicAdd(Result, SharedMem[0]);
    }
}

void Solve(const float *A, const float *B, float *Result, int N)
{
    int GridDim = (N + (BLOCK_DIM * COARSE_FACTOR) - 1) / (BLOCK_DIM * COARSE_FACTOR);
    KernelDotProduct02<<<GridDim, BLOCK_DIM>>>(A, B, Result, N);
}
#endif

int main()
{
}
