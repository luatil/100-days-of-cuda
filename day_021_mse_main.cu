/*
 * Calculates MSE
 *
 * Input:  predictions = [1.0, 2.0, 3.0, 4.0]
 * targets = [1.5, 2.5, 3.5, 4.5]
 * Output: mse = 0.25
 */


#include <hip/hip_runtime.h>
#include <stdio.h>

#ifndef SOLUTION
#define SOLUTION 3
#endif

#if SOLUTION == 1
__device__ float Square(float x)
{
    return x * x;
}
__global__ void mse_kernel(const float *predictions, const float *targets, float *mse, int N)
{
    float sum = 0.0f;
    for (int i = 0; i < N; i++)
    {
        sum += Square(targets[i] - predictions[i]);
    }
    *mse = sum / N;
}

// predictions, targets, mse are device pointers
void solve(const float *predictions, const float *targets, float *mse, int N)
{
    mse_kernel<<<1, 1>>>(predictions, targets, mse, N);
    cudaDeviceSynchronize();
}
#elif SOLUTION == 2
#define BLOCK_DIM 256
#define COARSE_FACTOR 4

__device__ float Square(float X)
{
    return X * X;
}

__global__ void MseKernel(const float *Pred, const float *Target, float *Mse, int N)
{
    __shared__ float SharedMem[BLOCK_DIM];

    int Tid = COARSE_FACTOR * BLOCK_DIM * blockIdx.x + threadIdx.x;
    int Tx = threadIdx.x;

    float Sum = 0.0f;
    for (int I = 0; I < COARSE_FACTOR; I++)
    {
        int Pos = Tid + BLOCK_DIM * I;
        if (Pos < N)
        {
            Sum += Square(Pred[Pos] - Target[Pos]) / N;
        }
    }
    SharedMem[Tx] = Sum;

    for (int Stride = BLOCK_DIM / 2; Stride > 0; Stride >>= 1)
    {
        __syncthreads();
        if (Tx < Stride)
        {
            SharedMem[Tx] += SharedMem[Tx + Stride];
        }
    }

    __syncthreads();
    if (Tx == 0)
    {
        atomicAdd(Mse, SharedMem[0]);
    }
}

void solve(const float *predictions, const float *targets, float *mse, int N)
{
    int GridDim = (N + (BLOCK_DIM * COARSE_FACTOR) - 1) / (BLOCK_DIM * COARSE_FACTOR);
    MseKernel<<<GridDim, BLOCK_DIM>>>(predictions, targets, mse, N);
    cudaDeviceSynchronize();
}
#elif SOLUTION == 3
#define BLOCK_DIM (256)
#define COARSE_FACTOR 4
#define WARP_SIZE 32

__device__ float Square(float X)
{
    return X * X;
}

__device__ __forceinline__ float WarpReduceSum(float Val)
{
    for (int Offset = WARP_SIZE / 2; Offset > 0; Offset /= 2)
    {
        Val += __shfl_down_sync(0xffffffff, Val, Offset);
    }
    return Val;
}

__device__ __forceinline__ float BlockReduceSum(float Val)
{
    static __shared__ float Shared[WARP_SIZE]; // Shared mem for 32 partial sums
    int Lane = threadIdx.x % WARP_SIZE;
    int Wid = threadIdx.x / WARP_SIZE;

    Val = WarpReduceSum(Val); // Each warp performs partial reduction

    if (Lane == 0)
        Shared[Wid] = Val; // Write reduced value to shared memory

    __syncthreads(); // Wait for all partial reductions

    // Read from shared memory only if that warp existed
    Val = (threadIdx.x < blockDim.x / WARP_SIZE) ? Shared[Lane] : 0;

    if (Wid == 0)
        Val = WarpReduceSum(Val); // Final reduce within first warp

    return Val;
}

__global__ void MseKernel(const float *Pred, const float *Target, float *Mse, int N)
{
    int Tid = COARSE_FACTOR * BLOCK_DIM * blockIdx.x + threadIdx.x;

    float Sum = 0.0f;
    for (int I = 0; I < COARSE_FACTOR; I++)
    {
        int Pos = Tid + BLOCK_DIM * I;
        if (Pos < N)
        {
            Sum += Square(Pred[Pos] - Target[Pos]) / N;
        }
    }

    Sum = BlockReduceSum(Sum);

    if (threadIdx.x == 0)
    {
        atomicAdd(Mse, Sum);
    }
}

void Solve(const float *Predictions, const float *Targets, float *Mse, int N)
{
    int GridDim = (N + (BLOCK_DIM * COARSE_FACTOR) - 1) / (BLOCK_DIM * COARSE_FACTOR);
    MseKernel<<<GridDim, BLOCK_DIM>>>(Predictions, Targets, Mse, N);
    hipDeviceSynchronize();
}
#elif SOLUTION == 4
#else
#endif

int main()
{
    float Pred[] = {1.0, 2.0f, 3.0f, 4.0f};
    float Tgt[] = {1.5, 2.5, 3.5, 4.5};
    float Eo = 0.25;

    int Sizeb = sizeof(Pred);

    float *DPred, *DTgt, *DMse;

    hipMalloc(&DPred, Sizeb);
    hipMalloc(&DTgt, Sizeb);
    hipMalloc(&DMse, sizeof(float));

    hipMemcpy(DPred, Pred, Sizeb, hipMemcpyHostToDevice);
    hipMemcpy(DTgt, Tgt, Sizeb, hipMemcpyHostToDevice);

    Solve(DPred, DTgt, DMse, sizeof(Pred) / sizeof(Pred[0]));

    float Mse = 0.0f;
    hipMemcpy(&Mse, DMse, sizeof(float), hipMemcpyDeviceToHost);

    printf("mse = %.3f\n", Mse);
    printf("eo = %.3f\n", Eo);
}
